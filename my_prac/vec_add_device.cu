#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 1024
#define CUDA_CHECK(x) \
    do { \
        hipError_t err = x; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

void init_vector(float *vec, int n) {
  for(int i=0; i<n; i++) {
    vec[i] = (float)rand() / RAND_MAX;
  }
}

__global__ void sum_vector(float *d_a, float *d_b, float *d_c, int n){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < n)
        d_c[i] = d_a[i] + d_b[i];
}

int main() {

    float *h_A, *h_B;
    float *d_A, *d_B ,*d_C;
    int size = N * sizeof(float);

    // alloting memory in cpu
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);

    // alloting memory in GPU
    CUDA_CHECK(hipMalloc(&d_B, size));
    CUDA_CHECK(hipMalloc(&d_C, size));
    CUDA_CHECK(hipMalloc(&d_A, size));
    // generating random values in vector
    init_vector(h_A, N);
    init_vector(h_B, N);
    
    // tranfering data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // calling kernel
    sum_vector <<< ceil(N/256.0), 256 >>> (d_A, d_B, d_C, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // freeing device memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}
